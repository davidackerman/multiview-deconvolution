/*
 * standardCUDAfunctions.cu
 *
 *  Created on: Jul 24, 2014
 *      Author: preibisch
 */
#include "book.h"
#include "hip/hip_runtime.h"
#include "standardCUDAfunctions.h"

//==============================================
int getCUDAcomputeCapabilityMajorVersion(int devCUDA)
{
	int major = 0, minor = 0;
	hipDeviceComputeCapability(&major, &minor,devCUDA);

	return major;
}
int getCUDAcomputeCapabilityMinorVersion(int devCUDA)
{
	int major = 0, minor = 0;
	hipDeviceComputeCapability(&major, &minor,devCUDA);

	return minor;
}

int getNumDevicesCUDA()
{
	int count = 0;
	HANDLE_ERROR(hipGetDeviceCount ( &count ));
	return count;
}
void getNameDeviceCUDA(int devCUDA, char* name)
{
	hipDeviceProp_t prop;
	HANDLE_ERROR( hipGetDeviceProperties(&prop, devCUDA));

	memcpy(name,prop.name,sizeof(char)*256);
}
long long int getMemDeviceCUDA(int devCUDA)
{
	hipDeviceProp_t prop;
	HANDLE_ERROR( hipGetDeviceProperties(&prop, devCUDA));
	return ((long long int)prop.totalGlobalMem);
}
