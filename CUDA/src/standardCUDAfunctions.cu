/*
 * standardCUDAfunctions.cu
 *
 *  Created on: Jul 24, 2014
 *      Author: preibisch
 */
#include "book.h"
#include "hip/hip_runtime.h"
#include "standardCUDAfunctions.h"

//==============================================
int getCUDAcomputeCapabilityMajorVersion(int devCUDA)
{
	int major = 0, minor = 0;
	hipDeviceComputeCapability(&major, &minor,devCUDA);

	return major;
}
int getCUDAcomputeCapabilityMinorVersion(int devCUDA)
{
	int major = 0, minor = 0;
	hipDeviceComputeCapability(&major, &minor,devCUDA);

	return minor;
}

int getNumDevicesCUDA()
{
	int count = 0;
	HANDLE_ERROR(hipGetDeviceCount ( &count ));
	return count;
}
void getNameDeviceCUDA(int devCUDA, char* name)
{
	hipDeviceProp_t prop;
	HANDLE_ERROR( hipGetDeviceProperties(&prop, devCUDA));

	memcpy(name,prop.name,sizeof(char)*256);
}

#include <iostream>

bool isDeviceCUDAusedByDisplay(int devCUDA)
{
	int has_timeout;
	HANDLE_ERROR( hipDeviceGetAttribute(&has_timeout, hipDeviceAttributeKernelExecTimeout, devCUDA) );
	//std::cout << has_timeout << std::endl;
	return (has_timeout > 0);
}

long long int getMemDeviceCUDA(int devCUDA)
{
	hipDeviceProp_t prop;
	HANDLE_ERROR( hipGetDeviceProperties(&prop, devCUDA));
	return ((long long int)prop.totalGlobalMem);
}

long long int getAvailableMemDeviceCUDA(int devCUDA)
{
	setDeviceCUDA(devCUDA);
	size_t free, total;
	HANDLE_ERROR(hipMemGetInfo(&free, &total));
	return ((long long int)free);
}

void setDeviceCUDA(int devCUDA)
{
	HANDLE_ERROR(hipSetDevice(devCUDA));	
}

void resetDeviceCUDA(int devCUDA)
{
	HANDLE_ERROR(hipSetDevice(devCUDA));
	HANDLE_ERROR(hipDeviceReset());
}