#include "hip/hip_runtime.h"
/*
* Copyright(C) 2015 by  Fernando Amat
* See license.txt for full license and copyright notice.
*
* Authors: Fernando Amat
*  multivieDeconvolution.cpp
*
*  Created on : June 5th, 2015
* Author : Fernando Amat
*
* \brief main interface to execute multiview deconvolution (it has ome abstract methods)
*/

#include <cstdint>
#include <iostream>
#include <algorithm>
#include <fstream>
#include "multiviewDeconvolution.h"
#include "book.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "commonCUDA.h"


using namespace std;

//WARNING: for cuFFT the fastest running index is z direction!!! so pos = z + imDim[2] * (y + imDim[1] * x)
template<class imageType>
__global__ void __launch_bounds__(MAX_THREADS_CUDA) fftShiftKernel(imageType* kernelCUDA, imageType* kernelPaddedCUDA, int kernelDim_0, int kernelDim_1, int kernelDim_2, int imDim_0, int imDim_1, int imDim_2)
{
	int kernelSize = kernelDim_0 * kernelDim_1 * kernelDim_2;

	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if (tid<kernelSize)
	{
		//find coordinates
		int64_t x, y, z, aux;
		z = tid % kernelDim_2;
		aux = (tid - z) / kernelDim_2;
		y = aux % kernelDim_1;
		x = (aux - y) / kernelDim_1;

		//center coordinates
		x -= kernelDim_0 / 2;
		y -= kernelDim_1 / 2;
		z -= kernelDim_2 / 2;

		//circular shift if necessary
		if (x<0) x += imDim_0;
		if (y<0) y += imDim_1;
		if (z<0) z += imDim_2;

		//calculate position in padded kernel
		aux = z + imDim_2 * (y + imDim_1 * x);

		//copy value
		kernelPaddedCUDA[aux] = kernelCUDA[tid];//for the most part it should be a coalescent access in both places
	}
}

////////////////////////////////////////////////////////////////////////////////
// Modulate Fourier image of padded data by Fourier image of padded kernel
// and normalize by FFT size
////////////////////////////////////////////////////////////////////////////////
//Adapted from CUDA SDK examples
__device__ void mulAndScale(hipfftComplex& a, const hipfftComplex& b, const float& c)
{
	hipfftComplex t = { c * (a.x * b.x - a.y * b.y), c * (a.y * b.x + a.x * b.y) };
	a = t;
};

//we multiply by conj(b) = {b.x, -b.y}
__device__ void mulAndScale_conj(hipfftComplex& a, const hipfftComplex& b, const float& c)
{
	hipfftComplex t = { c * (a.x * b.x + a.y * b.y), c * (a.y * b.x - a.x * b.y) };
	a = t;
};

__device__ hipfftComplex mulAndScale_outOfPlace(const hipfftComplex& a, const hipfftComplex& b, const float& c)
{
	return  { c * (a.x * b.x - a.y * b.y), c * (a.y * b.x + a.x * b.y) };	
};

__global__ void modulateAndNormalize_kernel(hipfftComplex *d_Dst, const hipfftComplex *d_Src, long long int dataSize, float c)
{
	std::int64_t i = (std::int64_t)blockDim.x * (std::int64_t)blockIdx.x + (std::int64_t)threadIdx.x;
	std::int64_t offset = (std::int64_t)blockDim.x * (std::int64_t)gridDim.x;
	while (i < dataSize)
	{
		//TODO: try speed difference without intermediate variables
		hipfftComplex a = d_Src[i];
		hipfftComplex b = d_Dst[i];
		mulAndScale(b, a, c);
		d_Dst[i] = b;

		i += offset;
	}
};

__global__ void modulateAndNormalize_conj_kernel(hipfftComplex *d_Dst, const hipfftComplex *d_Src, long long int dataSize, float c)
{
	std::int64_t i = (std::int64_t)blockDim.x * (std::int64_t)blockIdx.x + (std::int64_t)threadIdx.x;
	std::int64_t offset = (std::int64_t)blockDim.x * (std::int64_t)gridDim.x;
	while (i < dataSize)
	{
		//TODO: try speed difference without intermediate variables
		hipfftComplex a = d_Src[i];
		hipfftComplex b = d_Dst[i];
		mulAndScale_conj(b, a, c);
		d_Dst[i] = b;

		i += offset;
	}
};

__global__ void modulateAndNormalize_outOfPlace_kernel(hipfftComplex *d_Dst, const hipfftComplex *d_Src1, const hipfftComplex *d_Src2, long long int dataSize, float c)
{
	std::int64_t i = (std::int64_t)blockDim.x * (std::int64_t)blockIdx.x + (std::int64_t)threadIdx.x;
	std::int64_t offset = (std::int64_t)blockDim.x * (std::int64_t)gridDim.x;
	while (i < dataSize)
	{
		d_Dst[i] = mulAndScale_outOfPlace(d_Src1[i], d_Src2[i], c);
		i += offset;
	}
};
//===========================================================================

template<class imgType>
multiviewDeconvolution<imgType>::multiviewDeconvolution()
{
	J.resize(1);//allocate for the output	
	fftPlanInv = -1;
	fftPlanFwd = -1;
}

//=======================================================
template<class imgType>
multiviewDeconvolution<imgType>::~multiviewDeconvolution()
{
	if (fftPlanInv >= 0)
	{
		(hipfftDestroy(fftPlanInv)); HANDLE_ERROR_KERNEL;
	}
	if (fftPlanFwd >= 0)
	{
		(hipfftDestroy(fftPlanFwd)); HANDLE_ERROR_KERNEL;
	}
}


//=======================================================
template<class imgType>
void multiviewDeconvolution<imgType>::setNumberOfViews(int numViews)
{
	weights.resize(numViews);
	psf.resize(numViews);
	img.resize(numViews);
}

//=======================================================
template<class imgType>
int multiviewDeconvolution<imgType>::readImage(const std::string& filename, int pos, const std::string& type)
{
	if (type.compare("weight") == 0)
		return weights.readImage(filename, pos);
	else if (type.compare("psf") == 0)
		return psf.readImage(filename, pos);
	else if (type.compare("img") == 0)
		return img.readImage(filename, pos);
	
	cout << "ERROR: multiviewDeconvolution<imgType>::readImage :option " << type << " not recognized" << endl;
	return 3;
}

//=======================================================
template<class imgType>
void multiviewDeconvolution<imgType>::padArrayWithZeros(const std::uint32_t *dimsAfterPad, int pos, const std::string& type)
{
	if (type.compare("weight") == 0)
		return weights.padArrayWithZeros(pos, dimsAfterPad);
	else if (type.compare("psf") == 0)
		return psf.padArrayWithZeros(pos, dimsAfterPad);
	else if (type.compare("img") == 0)
		return img.padArrayWithZeros(pos, dimsAfterPad);

	cout << "ERROR: multiviewDeconvolution<imgType>::readImage :option " << type << " not recognized" << endl;	
}


//=======================================================
template<class imgType>
int multiviewDeconvolution<imgType>::readROI(const std::string& filename, int pos, const std::string& type, const klb_ROI& ROI)
{
	if (type.compare("weight") == 0)
		return weights.readROI(filename, pos, ROI);
	else if (type.compare("psf") == 0)
		return psf.readROI(filename, pos, ROI);
	else if (type.compare("img") == 0)
		return img.readROI(filename, pos, ROI);

	cout << "ERROR: multiviewDeconvolution<imgType>::readImage :option " << type << " not recognized" << endl;
	return 3;
}

//=======================================================
template<class imgType>
int multiviewDeconvolution<imgType>::allocate_workspace(imgType imgBackground)
{
	//const values throughout the function
	const bool useWeights = (weights.getPointer_CPU(0) != NULL);
	const int64_t nImg = img.numElements(0);
	const size_t nViews = img.getNumberOfViews();
	const int64_t imSizeFFT = nImg + (2 * img.dimsImgVec[0].dims[2] * img.dimsImgVec[0].dims[1]); //size of the R2C transform in cuFFTComple

	//variables needed for this function	
	psfType *psf_notPadded_GPU = NULL;//to store original PSF

	if (nViews == 0)
	{
		cout << "ERROR:multiviewDeconvolution<imgType>::allocate_workspace(): no views loaded to start process" << endl;
		return 2;
	}

	//allocate temporary mmeory to nromalize weights
	weightType *weightAvg_GPU = NULL;
	if (useWeights)
	{
		HANDLE_ERROR(hipMalloc((void**)&(weightAvg_GPU), nImg * sizeof(weightType)));
		HANDLE_ERROR(hipMemset(weightAvg_GPU, 0, nImg * sizeof(weightType)));
	}



	//preparing FFT plans
	hipfftPlan3d(&fftPlanFwd, img.dimsImgVec[0].dims[2], img.dimsImgVec[0].dims[1], img.dimsImgVec[0].dims[0], HIPFFT_R2C); HANDLE_ERROR_KERNEL;
	cufftSetCompatibilityMode(fftPlanFwd, CUFFT_COMPATIBILITY_NATIVE); HANDLE_ERROR_KERNEL; //for highest performance since we do not need FFTW compatibility
	hipfftPlan3d(&fftPlanInv, img.dimsImgVec[0].dims[2], img.dimsImgVec[0].dims[1], img.dimsImgVec[0].dims[0], HIPFFT_C2R); HANDLE_ERROR_KERNEL;
	cufftSetCompatibilityMode(fftPlanInv, CUFFT_COMPATIBILITY_NATIVE); HANDLE_ERROR_KERNEL;

	//allocate memory and precompute things for each view things for each vieww
	cout << "===================TODO: load img and weights on the fly to CPU to avoid consuming too much memory====================" << endl;
	for (size_t ii = 0; ii < nViews; ii++)
	{
		//load img for ii-th to CPU 
		//cout << "===================TODO: load weights on the fly to CPU to avoid consuming too much memory====================" << endl;
		//allocate memory for image in the GPU		
		img.allocateView_GPU(ii, nImg * sizeof(imgType));
		//transfer image
		HANDLE_ERROR(hipMemcpy(img.getPointer_GPU(ii), img.getPointer_CPU(ii), nImg * sizeof(imgType), hipMemcpyHostToDevice));
		//deallocate memory from CPU
		img.deallocateView_CPU(ii);
        //subtract background
		if ( imgBackground > 0)
			elementwiseOperationInPlace<imgType>(img.getPointer_GPU(ii), imgBackground, nImg, op_elementwise_type::minus_positive);

		if (useWeights)
		{
			//load weights for ii-th to CPU 
			//cout << "===================TODO: load weights on the fly to CPU to avoid consuming too much memory====================" << endl;
			//allocate memory for weights in the GPU			
			weights.allocateView_GPU(ii, nImg * sizeof(weightType));
			//transfer image
			HANDLE_ERROR(hipMemcpy(weights.getPointer_GPU(ii), weights.getPointer_CPU(ii), nImg * sizeof(weightType), hipMemcpyHostToDevice));
			//deallocate memory from CPU
			weights.deallocateView_CPU(ii);

			//call kernel to update weightAvg_GPU
			elementwiseOperationInPlace<weightType>(weightAvg_GPU, weights.getPointer_GPU(ii), nImg, op_elementwise_type::plus);
		}

		//allocate memory for PSF FFT
		const int64_t psfSize = psf.numElements(ii);
		HANDLE_ERROR(hipMalloc((void**)&(psf_notPadded_GPU), (psfSize) * sizeof(psfType)));		
		psf.allocateView_GPU(ii, imSizeFFT * sizeof(psfType));

		//transfer psf
		HANDLE_ERROR(hipMemcpy(psf_notPadded_GPU, psf.getPointer_CPU(ii), psfSize * sizeof(psfType), hipMemcpyHostToDevice));

		//apply ffshift to kernel and pad it with zeros so we can calculate convolution with FFT
		int numThreads = std::min((int64_t)MAX_THREADS_CUDA/4, psfSize);
		int numBlocks = std::min((int64_t)MAX_BLOCKS_CUDA, (int64_t)(psfSize + (int64_t)(numThreads - 1)) / ((int64_t)numThreads));
		HANDLE_ERROR(hipMemset(psf.getPointer_GPU(ii), 0, imSizeFFT * sizeof(psfType)));		
		fftShiftKernel << <numBlocks, numThreads >> >(psf_notPadded_GPU, psf.getPointer_GPU(ii), psf.dimsImgVec[ii].dims[2], psf.dimsImgVec[ii].dims[1], psf.dimsImgVec[ii].dims[0], img.dimsImgVec[ii].dims[2], img.dimsImgVec[ii].dims[1], img.dimsImgVec[ii].dims[0]); HANDLE_ERROR_KERNEL;


#ifdef _DEBUG
		//char buffer[256];
		//sprintf(buffer, "E:/temp/deconvolution/PSFpadded_view%d.raw", ii);
		//debug_writeGPUarray(psf.getPointer_GPU(ii), img.dimsImgVec[0], string(buffer));		
#endif

		//execute FFT.  If idata and odata are the same, this method does an in-place transform
		hipfftExecR2C(fftPlanFwd, psf.getPointer_GPU(ii), (hipfftComplex *)(psf.getPointer_GPU(ii))); HANDLE_ERROR_KERNEL;

		//release memory for PSF
		HANDLE_ERROR(hipFree(psf_notPadded_GPU));
		psf.deallocateView_CPU(ii);
	}	
	
	
	if (useWeights)
	{
		cout << "======TODO: during normalization check elements with all zero weights====" << endl;
		//normalize weights	
		for (size_t ii = 0; ii < nViews; ii++)
		{
			elementwiseOperationInPlace(weights.getPointer_GPU(ii), weightAvg_GPU, nImg, op_elementwise_type::divide);
		}

		//deallocate temporary memory to nromalize weights
		HANDLE_ERROR(hipFree(weightAvg_GPU));
		weightAvg_GPU = NULL; 
	}


	//allocate memory for final result
	J.resize(1);
	J.setImgDims(0, img.dimsImgVec[0]);
	J.allocateView_GPU(0, nImg * sizeof(outputType));
	J.allocateView_CPU(0, nImg );

	//initialize final results as weighted average of all views
	HANDLE_ERROR(hipMemset(J.getPointer_GPU(0), 0, nImg * sizeof(outputType)));
	for (size_t ii = 0; ii < nViews; ii++)
	{
		elementwiseOperationOutOfPlace(J.getPointer_GPU(0), weights.getPointer_GPU(ii), img.getPointer_GPU(ii), nImg, op_elementwise_type::compound_plus);
	}


	return 0;
}
//=======================================================
template<class imgType>
int multiviewDeconvolution<imgType>::allocate_workspace_update_multiGPU(imgType imgBackground, bool useWeights)
{
	//const values throughout the function	
	const int64_t nImg = img.numElements(0);
	const size_t nViews = img.getNumberOfViews();
	const int64_t imSizeFFT = nImg + (2 * img.dimsImgVec[0].dims[2] * img.dimsImgVec[0].dims[1]); //size of the R2C transform in cuFFTComple

	
	if (nViews == 0)
	{
		cout << "ERROR:multiviewDeconvolution<imgType>::allocate_workspace(): no views loaded to start process" << endl;
		return 2;
	}

	//allocate temporary memory to nromalize weights
	weightType *weightAvg_GPU = NULL;
	if (useWeights)
	{
		HANDLE_ERROR(hipMalloc((void**)&(weightAvg_GPU), nImg * sizeof(weightType)));
		HANDLE_ERROR(hipMemset(weightAvg_GPU, 0, nImg * sizeof(weightType)));
	}

	
	//allocate memory and precompute things for each view things for each vieww	
	for (size_t ii = 0; ii < nViews; ii++)
	{
		//load img for ii-th to CPU 
		//cout << "===================TODO: load weights on the fly to CPU to avoid consuming too much memory====================" << endl;
		//allocate memory for image in the GPU		
		//img.allocateView_GPU(ii, nImg * sizeof(imgType)); memory has already been allocate in the init phase
		//transfer image
		HANDLE_ERROR(hipMemcpy(img.getPointer_GPU(ii), img.getPointer_CPU(ii), nImg * sizeof(imgType), hipMemcpyHostToDevice));
		//deallocate memory from CPU
		img.deallocateView_CPU(ii);
		//subtract background
		if (imgBackground > 0)
			elementwiseOperationInPlace<imgType>(img.getPointer_GPU(ii), imgBackground, nImg, op_elementwise_type::minus_positive);

		if (useWeights)
		{
			//load weights for ii-th to CPU 
			//cout << "===================TODO: load weights on the fly to CPU to avoid consuming too much memory====================" << endl;			
			//transfer image
			HANDLE_ERROR(hipMemcpy(weights.getPointer_GPU(ii), weights.getPointer_CPU(ii), nImg * sizeof(weightType), hipMemcpyHostToDevice));
			//deallocate memory from CPU
			weights.deallocateView_CPU(ii);

			//call kernel to update weightAvg_GPU
			elementwiseOperationInPlace<weightType>(weightAvg_GPU, weights.getPointer_GPU(ii), nImg, op_elementwise_type::plus);
		}
		
	}


	if (useWeights)
	{
		cout << "======TODO: during normalization check elements with all zero weights====" << endl;
		//normalize weights	
		for (size_t ii = 0; ii < nViews; ii++)
		{
			elementwiseOperationInPlace(weights.getPointer_GPU(ii), weightAvg_GPU, nImg, op_elementwise_type::divide);
		}

		//deallocate temporary memory to nromalize weights
		HANDLE_ERROR(hipFree(weightAvg_GPU));
		weightAvg_GPU = NULL;
	}
	
	//initialize final results as weighted average of all views
	HANDLE_ERROR(hipMemset(J.getPointer_GPU(0), 0, nImg * sizeof(outputType)));
	for (size_t ii = 0; ii < nViews; ii++)
	{
		elementwiseOperationOutOfPlace(J.getPointer_GPU(0), weights.getPointer_GPU(ii), img.getPointer_GPU(ii), nImg, op_elementwise_type::compound_plus);
	}


	return 0;
}
//=======================================================
template<class imgType>
int multiviewDeconvolution<imgType>::allocate_workspace_init_multiGPU(const uint32_t blockDims[MAX_DATA_DIMS], bool useWeights)
{

	//const values throughout the function			
	const size_t nViews = psf.getNumberOfViews();	
	int64_t nImg = 1;
	for (int ii = 0; ii < MAX_DATA_DIMS; ii++)
		nImg *= blockDims[ii];

	const int64_t imSizeFFT = nImg + (2 * blockDims[2] * blockDims[1]); //size of the R2C transform in cuFFTComple

	//variables needed for this function	
	psfType *psf_notPadded_GPU = NULL;//to store original PSF

	if (nViews == 0)
	{
		cout << "ERROR:multiviewDeconvolution<imgType>::allocate_workspace(): no views loaded to start process" << endl;
		return 2;
	}
    

	//preparing FFT plans
	hipfftPlan3d(&fftPlanFwd, blockDims[2], blockDims[1], blockDims[0], HIPFFT_R2C); HANDLE_ERROR_KERNEL;
	cufftSetCompatibilityMode(fftPlanFwd, CUFFT_COMPATIBILITY_NATIVE); HANDLE_ERROR_KERNEL; //for highest performance since we do not need FFTW compatibility
	hipfftPlan3d(&fftPlanInv, blockDims[2], blockDims[1], blockDims[0], HIPFFT_C2R); HANDLE_ERROR_KERNEL;
	cufftSetCompatibilityMode(fftPlanInv, CUFFT_COMPATIBILITY_NATIVE); HANDLE_ERROR_KERNEL;


	//allocate memory and precompute things for each view things for each vieww	
	for (size_t ii = 0; ii < nViews; ii++)
	{
		//load img for ii-th to CPU 
		//cout << "===================TODO: load weights on the fly to CPU to avoid consuming too much memory====================" << endl;
		//allocate memory for image in the GPU		
		img.allocateView_GPU(ii, nImg * sizeof(imgType));
        //we do not have anything to upload yet		
		
		if (useWeights)
		{
			//load weights for ii-th to CPU 
			//cout << "===================TODO: load weights on the fly to CPU to avoid consuming too much memory====================" << endl;
			//allocate memory for weights in the GPU			
			weights.allocateView_GPU(ii, nImg * sizeof(weightType));
			
		}

		//allocate memory for PSF FFT
		const int64_t psfSize = psf.numElements(ii);
		HANDLE_ERROR(hipMalloc((void**)&(psf_notPadded_GPU), (psfSize)* sizeof(psfType)));
		psf.allocateView_GPU(ii, imSizeFFT * sizeof(psfType));

		//transfer psf
		HANDLE_ERROR(hipMemcpy(psf_notPadded_GPU, psf.getPointer_CPU(ii), psfSize * sizeof(psfType), hipMemcpyHostToDevice));

		//apply ffshift to kernel and pad it with zeros so we can calculate convolution with FFT
		int numThreads = std::min((int64_t)MAX_THREADS_CUDA / 4, psfSize);
		int numBlocks = std::min((int64_t)MAX_BLOCKS_CUDA, (int64_t)(psfSize + (int64_t)(numThreads - 1)) / ((int64_t)numThreads));
		HANDLE_ERROR(hipMemset(psf.getPointer_GPU(ii), 0, imSizeFFT * sizeof(psfType)));
		fftShiftKernel << <numBlocks, numThreads >> >(psf_notPadded_GPU, psf.getPointer_GPU(ii), psf.dimsImgVec[ii].dims[2], psf.dimsImgVec[ii].dims[1], psf.dimsImgVec[ii].dims[0], blockDims[2], blockDims[1], blockDims[0]); HANDLE_ERROR_KERNEL;


		//execute FFT.  If idata and odata are the same, this method does an in-place transform
		hipfftExecR2C(fftPlanFwd, psf.getPointer_GPU(ii), (hipfftComplex *)(psf.getPointer_GPU(ii))); HANDLE_ERROR_KERNEL;

		//release memory for PSF
		HANDLE_ERROR(hipFree(psf_notPadded_GPU));
		psf.deallocateView_CPU(ii);
	}	

	//allocate memory for final result
	J.resize(1);
	dimsImg aux; 
    aux.ndims = MAX_DATA_DIMS;
	for (int ii = 0; ii < MAX_DATA_DIMS; ii++)
		aux.dims[ii] = blockDims[ii];
	J.setImgDims(0, aux);
	J.allocateView_GPU(0, nImg * sizeof(outputType));
	J.allocateView_CPU(0, nImg);	


	return 0;
}

//=======================================================
template<class imgType>
void multiviewDeconvolution<imgType>::deconvolution_LR_TV(int numIters, float lambdaTV)
{
	const bool useWeights = (weights.getPointer_CPU(0) != NULL);
	const int64_t nImg = img.numElements(0);
	const size_t nViews = img.getNumberOfViews();
	const int64_t imSizeFFT = nImg + (2 * img.dimsImgVec[0].dims[2] * img.dimsImgVec[0].dims[1]); //size of the R2C transform in cuFFTComple

	int numThreads = std::min((std::int64_t)MAX_THREADS_CUDA/4, imSizeFFT / 2);//we are using complex numbers
	int numBlocks = std::min((std::int64_t)MAX_BLOCKS_CUDA, (std::int64_t)(imSizeFFT / 2 + (std::int64_t)(numThreads - 1)) / ((std::int64_t)numThreads));    

	//allocate extra memory required for intermediate calculations
	outputType *J_GPU_FFT, *aux_FFT, *aux_LR;
	outputType *TV_GPU = NULL;
	HANDLE_ERROR(hipMalloc((void**)&(J_GPU_FFT), imSizeFFT * sizeof(outputType)));//for J FFT
	HANDLE_ERROR(hipMalloc((void**)&(aux_FFT), imSizeFFT * sizeof(outputType)));//to hold products between FFT
	HANDLE_ERROR(hipMalloc((void**)&(aux_LR), nImg * sizeof(outputType)));//to hold LR update 

	//loop for each iteration
	for (int iter = 0; iter < numIters; iter++)
	{
		//copy current solution
		elementwiseOperationInPlace(J_GPU_FFT, J.getPointer_GPU(0), nImg, op_elementwise_type::copy);
		//precompute FFT for current solution
		hipfftExecR2C(fftPlanFwd, J_GPU_FFT, (hipfftComplex *)J_GPU_FFT); HANDLE_ERROR_KERNEL;

		//precalculate TV on J
		if (lambdaTV > 0)
		{
			cout << "==============TODO: calculate total variation==================" << endl;
		}

		//reset update
		HANDLE_ERROR(hipMemset(aux_LR, 0, nImg * sizeof(outputType)));
		//main loop over the different views
		for (int vv = 0; vv < nViews; vv++)
		{
			//multiply LR currant result and kernel in fourier space (and normalize)
			//NOTE: from CUFFT manual: CUFFT performs un-normalized FFTs; that is, performing a forward FFT on an input data set followed by an inverse FFT on the resulting set yields data that is equal to the input scaled by the number of elements.			
			modulateAndNormalize_outOfPlace_kernel << <numBlocks, numThreads >> >((hipfftComplex *)(aux_FFT), (hipfftComplex *)(J_GPU_FFT), (hipfftComplex *)(psf.getPointer_GPU(vv)), imSizeFFT / 2, 1.0f / (float)(nImg));//last parameter is the size of the FFT

			//inverse FFT 
			hipfftExecC2R(fftPlanInv, (hipfftComplex *)aux_FFT, aux_FFT); HANDLE_ERROR_KERNEL;


#ifdef _DEBUG
			char buffer[256];
			sprintf(buffer, "E:/temp/deconvolution/J_iter%.4d.raw", iter);
			if (vv == 0)
				debug_writeGPUarray(J.getPointer_GPU(0), J.dimsImgVec[0], string(buffer));
			/*            
			sprintf(buffer, "E:/temp/deconvolution/img_view%.4d.raw", vv);
			if ( iter == 0 )
				debug_writeGPUarray(img.getPointer_GPU(0), img.dimsImgVec[0], string(buffer));
			sprintf(buffer, "E:/temp/deconvolution/weights_view%.4d.raw", vv);
			if (iter == 0)
				debug_writeGPUarray(weights.getPointer_GPU(0), img.dimsImgVec[0], string(buffer));			
			sprintf(buffer, "E:/temp/deconvolution/JconvPSF_iter%.4d_view%d.raw", iter, vv);
			debug_writeGPUarray(aux_FFT, J.dimsImgVec[0], string(buffer));
			sprintf(buffer, "E:/temp/deconvolution/JFFT_iter%.4d.raw", iter);
			debug_writeGPUarray(J_GPU_FFT, J.dimsImgVec[0], string(buffer));
			sprintf(buffer, "E:/temp/deconvolution/PSFpaddedFfft_iter%.4d_view%d.raw", iter, vv);
			debug_writeGPUarray(psf.getPointer_GPU(vv), J.dimsImgVec[0], string(buffer));			
            */
#endif

			//calculate ratio img.getPointer_GPU(ii) ./ aux_FFT
			elementwiseOperationInPlace(aux_FFT, img.getPointer_GPU(vv), nImg, op_elementwise_type::divide_inv);

			//calculate FFT of ratio (for convolution)
			hipfftExecR2C(fftPlanFwd, aux_FFT, (hipfftComplex *)aux_FFT); HANDLE_ERROR_KERNEL;

			//multiply auxFFT and FFT(PSF)*
			modulateAndNormalize_conj_kernel << <numBlocks, numThreads >> >((hipfftComplex *)(aux_FFT), (hipfftComplex *)(psf.getPointer_GPU(vv)), imSizeFFT / 2, 1.0f / (float)(nImg));

			//inverse FFT
			hipfftExecC2R(fftPlanInv, (hipfftComplex *)aux_FFT, aux_FFT); HANDLE_ERROR_KERNEL;

			//add the value
			if (useWeights)
			{
				elementwiseOperationOutOfPlace(aux_LR, weights.getPointer_GPU(vv), aux_FFT, nImg, op_elementwise_type::compound_multiply);
			}
			else{
				elementwiseOperationInPlace(aux_LR, aux_FFT, nImg, op_elementwise_type::plus);
			}
			
		}

		//normalize weights if we are just using averaging
		if (!useWeights)
			elementwiseOperationInPlace(aux_LR, 1.0f / (float)nViews, nImg, op_elementwise_type::multiply);		

		//apply TV
		if (lambdaTV > 0)
		{
			elementwiseOperationInPlace(aux_LR, TV_GPU, nImg, op_elementwise_type::divide);
		}

		//update LR 
		elementwiseOperationInPlace(J.getPointer_GPU(0), aux_LR, nImg, op_elementwise_type::multiply);
		
	}

	//release memory
	HANDLE_ERROR(hipFree(aux_LR));
	HANDLE_ERROR(hipFree(aux_FFT));
	HANDLE_ERROR(hipFree(J_GPU_FFT));
	if ( TV_GPU != NULL)
		HANDLE_ERROR(hipFree(TV_GPU));
	
}

//===========================================================
template<class imgType>
void multiviewDeconvolution<imgType>::debug_writDeconvolutionResultRaw(const std::string& filename)
{
	cout << "======DEBUGGING:multiviewDeconvolution<imgType>::debug_writDeconvolutionResultRaw========" << endl;
	cout << " writing raw file " << J.dimsImgVec[0].dims[0] << "x" << J.dimsImgVec[0].dims[1] << "x" << J.dimsImgVec[0].dims[2] << "x" << " in float to " << filename << endl;

	ofstream fid(filename.c_str(), ios::binary);

	fid.write((char*)(J.getPointer_CPU(0)), J.numElements(0) * sizeof(imgType)); 
	fid.close();
}

//===========================================================
template<class imgType>
void multiviewDeconvolution<imgType>::debug_writeGPUarray(float* ptr_GPU, dimsImg& dims, const std::string& filename)
{
	cout << "======DEBUGGING:multiviewDeconvolution<imgType>::debug_writeGPUarray========" << endl;
	cout << " writing raw file ";
	int64_t numElements = 1;

	for (int ii = 0; ii < dims.ndims; ii++)
	{
		numElements *= dims.dims[ii];
		cout << dims.dims[ii] << "x";
	}
	cout << " in float format to " << filename << endl;


	float* ptr_CPU = new float[numElements];
	HANDLE_ERROR(hipMemcpy(ptr_CPU, ptr_GPU, numElements * sizeof(float), hipMemcpyDeviceToHost));

	ofstream fid(filename.c_str(), ios::binary);
	fid.write((char*)(ptr_CPU), numElements * sizeof(float));
	fid.close();

	delete[] ptr_CPU;
}

//===========================================================
template<class imgType>
void multiviewDeconvolution<imgType>::debug_writeCPUarray(float* ptr_CPU, dimsImg& dims, const std::string& filename)
{
	cout << "======DEBUGGING:multiviewDeconvolution<imgType>::debug_writeCPUarray========" << endl;
	cout << " writing raw file ";
	int64_t numElements = 1;

	for (int ii = 0; ii < dims.ndims; ii++)
	{
		numElements *= dims.dims[ii];
		cout << dims.dims[ii] << "x";
	}
	cout << " in float format to " << filename << endl;


	
	ofstream fid(filename.c_str(), ios::binary);
	fid.write((char*)(ptr_CPU), numElements * sizeof(float));
	fid.close();

	
}


//=====================================================================
//WARNING: for cuFFT the fastest running index is z direction!!! so pos = z + imDim[2] * (y + imDim[1] * x)
//NOTE: to avoid transferring a large padded kernel, since memcpy is a limiting factor 

//uncomment to save intermediate steps
#define DEBUG_FFT_INTERMEDIATE_STEPS

template<class imgType>
imgType* multiviewDeconvolution<imgType>::convolution3DfftCUDA(const imgType* im, const std::int64_t* imDim, const imgType* kernel, const std::int64_t* kernelDim, int devCUDA)
{
	imgType* convResult = NULL;
	imgType* imCUDA = NULL;
	imgType* kernelCUDA = NULL;
	imgType* kernelPaddedCUDA = NULL;

	int dimsImage = 3;

	hipfftHandle fftPlanFwd, fftPlanInv;

#ifdef DEBUG_FFT_INTERMEDIATE_STEPS
	string filepath("E:/temp/deconvolution/");
#endif

	HANDLE_ERROR(hipSetDevice(devCUDA));

	long long int imSize = 1;
	long long int kernelSize = 1;
	for (int ii = 0; ii<dimsImage; ii++)
	{
		imSize *= (long long int) (imDim[ii]);
		kernelSize *= (long long int) (kernelDim[ii]);
	}

	long long int imSizeFFT = imSize + (long long int)(2 * imDim[0] * imDim[1]); //size of the R2C transform in cuFFTComplex

	//allocate memory for output result
	convResult = new imgType[imSize];

	//allocat ememory in GPU
	HANDLE_ERROR(hipMalloc((void**)&(imCUDA), imSizeFFT*sizeof(imgType)));//a little bit larger to allow in-place FFT
	HANDLE_ERROR(hipMalloc((void**)&(kernelCUDA), (kernelSize)*sizeof(imgType)));
	HANDLE_ERROR(hipMalloc((void**)&(kernelPaddedCUDA), imSizeFFT*sizeof(imgType)));


	//TODO: pad image to a power of 2 size in all dimensions (use whatever  boundary conditions you want to apply)
	//TODO: pad kernel to image size
	//TODO: pad kernel and image to xy(z/2 + 1) for in-place transform
	//NOTE: in the example for 2D convolution using FFT in the Nvidia SDK they do the padding in the GPU, but in might be pushing the memory in the GPU for large images.

	//printf("Copying memory (kernel and image) to GPU\n");
	HANDLE_ERROR(hipMemcpy(kernelCUDA, kernel, kernelSize*sizeof(imgType), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(imCUDA, im, imSize*sizeof(imgType), hipMemcpyHostToDevice));

	//apply ffshift to kernel and pad it with zeros so we can calculate convolution with FFT
	HANDLE_ERROR(hipMemset(kernelPaddedCUDA, 0, imSizeFFT*sizeof(imgType)));
	int numThreads = std::min((long long int)MAX_THREADS_CUDA, kernelSize);
	int numBlocks = std::min((long long int)MAX_BLOCKS_CUDA, (long long int)(kernelSize + (long long int)(numThreads - 1)) / ((long long int)numThreads));
	fftShiftKernel << <numBlocks, numThreads >> >(kernelCUDA, kernelPaddedCUDA, kernelDim[0], kernelDim[1], kernelDim[2], imDim[0], imDim[1], imDim[2]); HANDLE_ERROR_KERNEL;


	//make sure GPU finishes before we launch two different streams
	HANDLE_ERROR(hipDeviceSynchronize());

#ifdef DEBUG_FFT_INTERMEDIATE_STEPS
	dimsImg auxDimsImg;
	auxDimsImg.ndims = dimsImage;
	for (int ii = 0; ii < dimsImage; ii++)
		auxDimsImg.dims[ii] = imDim[dimsImage-1-ii];//flip coordinates

	debug_writeGPUarray(kernelPaddedCUDA, auxDimsImg, string(filepath + "cudafft3d_kernelPaddedCuda.raw"));
#endif

	//printf("Creating R2C & C2R FFT plans for size %i x %i x %i\n",imDim[0],imDim[1],imDim[2]);
	hipfftPlan3d(&fftPlanFwd, imDim[0], imDim[1], imDim[2], HIPFFT_R2C); HANDLE_ERROR_KERNEL;
	cufftSetCompatibilityMode(fftPlanFwd, CUFFT_COMPATIBILITY_NATIVE); HANDLE_ERROR_KERNEL; //for highest performance since we do not need FFTW compatibility
	hipfftPlan3d(&fftPlanInv, imDim[0], imDim[1], imDim[2], HIPFFT_C2R); HANDLE_ERROR_KERNEL;
	cufftSetCompatibilityMode(fftPlanInv, CUFFT_COMPATIBILITY_NATIVE); HANDLE_ERROR_KERNEL;

	//transforming convolution kernel; TODO: if I do multiple convolutions with the same kernel I could reuse the results at teh expense of using out-of place memory (and then teh layout of the data is different!!!! so imCUDAfft should also be out of place)
	//NOTE: from CUFFT manual: If idata and odata are the same, this method does an in-place transform.
	//NOTE: from CUFFT manual: inplace output data xy(z/2 + 1) with fcomplex. Therefore, in order to perform an in-place FFT, the user has to pad the input array in the last dimension to Nn2 + 1 complex elements interleaved. Note that the real-to-complex transform is implicitly forward.
	hipfftExecR2C(fftPlanFwd, imCUDA, (hipfftComplex *)imCUDA); HANDLE_ERROR_KERNEL;
	//transforming image
	hipfftExecR2C(fftPlanFwd, kernelPaddedCUDA, (hipfftComplex *)kernelPaddedCUDA); HANDLE_ERROR_KERNEL;

#ifdef DEBUG_FFT_INTERMEDIATE_STEPS
	debug_writeGPUarray(kernelPaddedCUDA, auxDimsImg, string(filepath + "cudafft3d_kernelPaddedCuda_fft.raw"));
	debug_writeGPUarray(imCUDA, auxDimsImg, string(filepath + "cudafft3d_im_fft.raw"));
#endif


	//multiply image and kernel in fourier space (and normalize)
	//NOTE: from CUFFT manual: CUFFT performs un-normalized FFTs; that is, performing a forward FFT on an input data set followed by an inverse FFT on the resulting set yields data that is equal to the input scaled by the number of elements.
	numThreads = std::min((long long int)MAX_THREADS_CUDA, imSizeFFT / 2);//we are using complex numbers
	numBlocks = std::min((long long int)MAX_BLOCKS_CUDA, (long long int)(imSizeFFT / 2 + (long long int)(numThreads - 1)) / ((long long int)numThreads));
	modulateAndNormalize_kernel << <numBlocks, numThreads >> >((hipfftComplex *)imCUDA, (hipfftComplex *)kernelPaddedCUDA, imSizeFFT / 2, 1.0f / (float)(imSize));//last parameter is the size of the FFT

#ifdef DEBUG_FFT_INTERMEDIATE_STEPS	
	debug_writeGPUarray(imCUDA, auxDimsImg, string(filepath + "cudafft3d_imTimesPSF_fft.raw"));
#endif

	//inverse FFT 
	hipfftExecC2R(fftPlanInv, (hipfftComplex *)imCUDA, imCUDA); HANDLE_ERROR_KERNEL;

	//copy result to host
	HANDLE_ERROR(hipMemcpy(convResult, imCUDA, sizeof(imgType)*imSize, hipMemcpyDeviceToHost));

	//release memory
	(hipfftDestroy(fftPlanInv)); HANDLE_ERROR_KERNEL;
	(hipfftDestroy(fftPlanFwd)); HANDLE_ERROR_KERNEL;
	HANDLE_ERROR(hipFree(imCUDA));
	HANDLE_ERROR(hipFree(kernelCUDA));
	HANDLE_ERROR(hipFree(kernelPaddedCUDA));

	return convResult;
}

//=================================================================
template<class imgType>
imgType* multiviewDeconvolution<imgType>::convolution3DfftCUDA_img_psf(size_t pos, int devCUDA)
{ 
	//we need to flip dimensions because cuFFT the fastest running dimension is the last one
	int64_t dimsI[3], dimsP[3];

	dimsI[0] = img.dimsImgVec[pos].dims[2];
	dimsI[1] = img.dimsImgVec[pos].dims[1];
	dimsI[2] = img.dimsImgVec[pos].dims[0];

	dimsP[0] = psf.dimsImgVec[pos].dims[2];
	dimsP[1] = psf.dimsImgVec[pos].dims[1];
	dimsP[2] = psf.dimsImgVec[pos].dims[0];

	imgType* aux = convolution3DfftCUDA(img.getPointer_CPU(pos), dimsI, psf.getPointer_CPU(pos), dimsP, devCUDA);
	//imgType* aux = convolution3DfftCUDA(img.getPointer_CPU(pos), img.dimsImgVec[pos].dims, psf.getPointer_CPU(pos), psf.dimsImgVec[pos].dims, devCUDA); 

	return aux;
};

//=================================================================
//declare all possible instantitation for the template
//TODO: right now the code can only handle float images since the rest of operations are carried in float point
//template class multiviewDeconvolution<uint16_t>;
//template class multiviewDeconvolution<uint8_t>;
template class multiviewDeconvolution<float>;